
#include <hip/hip_runtime.h>
__global__ void per_row_kernel(int* in,int N)
{
    int blockId = blockIdx.x * blockDim.y + threadIdx.y;
    int rn = blockDim.x * blockId + threadIdx.x;
    if(rn >= N)
      return; 
    int i;
    for(i = 0;i < N;i++)
    {
        int old_ind = N*rn + i;
        int new_ind = N*i + rn;
        if(rn < i)
        {
            int t = in[old_ind];
            in[old_ind] = in[new_ind];
            in[new_ind] = t;
        }
    }
}

__global__ void per_element_kernel(int *in,int N)
{
    int blockId = blockIdx.x + gridDim.x * (blockIdx.y + (gridDim.y * blockIdx.z));
    int ind = blockId * blockDim.x + threadIdx.x;
    int x = ind / N;
    int y = ind % N;
    if(x < y)
    {
        int t = in[N*x + y];
        in[N*x + y] = in[N*y + x];
        in[N*y + x] = t;
    }
}

__global__ void per_element_kernel_2D(int* in,int N)
{
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int ind = blockDim.x *((blockId * blockDim.y) + threadIdx.y) + threadIdx.x;
    if(ind >= N*N)
      return;
    int x = ind / N;
    int y = ind % N;
    if(x < y)
    {
        int t = in[N*x + y];
        in[N*x + y] = in[N*y + x];
        in[N*y + x] = t;
    }

}
