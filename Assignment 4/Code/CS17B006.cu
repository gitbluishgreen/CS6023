#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#define MAX 10000

struct minhelper
{
  int _a;
  minhelper(int a): _a(a){}
  __host__ __device__ int operator()(const int x,const int y)const{
    return min(MAX*max(_a,x/MAX)+x%MAX,MAX*max(_a,y/MAX)+y%MAX);
  }
};

int schedule(int N,int M,int* arrival_times,int* burst_times,int** cores_schedules,int* cs_lengths)
{
  thrust::device_vector<int> mini_time(M);
  thrust::sequence(thrust::device,mini_time.begin(),mini_time.end(),0);
  int i;
  for(i = 0;i < M;i++)
  {
      cores_schedules[i] = (int*)malloc(sizeof(int)*N);
      cs_lengths[i] = 0;
  }
  int turnaround_time = 0;
  int* x = (int*)malloc(sizeof(int));
  for(i = 0;i < N;i++)
  {
    int at = arrival_times[i];
    int b = burst_times[i];
    int ind1 = thrust::reduce(mini_time.begin(),mini_time.end(),(int)1000000000,minhelper(at));
    int ind = ind1 % MAX;
    turnaround_time += max((ind1 / MAX) - at,0) + b;
    cores_schedules[ind][cs_lengths[ind]] = i;
    cs_lengths[ind]++;
    int* x1 = thrust::raw_pointer_cast(&mini_time[ind]);
    *x = ind + (ind1/MAX + b) * MAX;
    hipMemcpy(x1,x,sizeof(int),hipMemcpyHostToDevice);
  }
  return turnaround_time;
}
