//-lm to link math.h - optional
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <unistd.h>
#include <fcntl.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <ctype.h>
#include <math.h>
__global__ void database_update(int* mat,int* col_comp,int* col_vals,int* upd_vals,int m,int n,int q)
{
    int gid = blockDim.x * blockIdx.x + threadIdx.x;
    if(gid >= m*q*20)
      return;
    int pn = gid/(m*q);
    int z = gid % (m*q);
    int qn = z%q;
    int rn = z/q;
    int col_c = col_comp[2*qn];
    int v = col_comp[2*qn+1];
    int col_upd = col_vals[qn*20+pn];
    int upd_val = upd_vals[qn*20+pn];
    if((mat[n*rn + (col_c-1)] == v) && (col_upd > 0))
      atomicAdd(&mat[n*rn + col_upd-1],upd_val);
}
int main(int argc,char* argv[])
{
    FILE* fp1 = fopen(argv[1],"r");
    FILE* fp2 = fopen(argv[2],"w");
    int m,n,q;
    fscanf(fp1,"%d%d",&m,&n);
    int* arr = (int*)malloc(sizeof(int)*m*n);
    int i,j;
    for(i = 0;i < m;i++)
    {
        for(j = 0;j < n;j++)
        {
            fscanf(fp1,"%d",&arr[i*n + j]);
        }
    }
    fscanf(fp1,"%d",&q);
    int* upd_vals = (int*)malloc(sizeof(int)*q*20);//worst case size of the update array
    int* col_vals = (int*)malloc(sizeof(int)*q*20);
    int* comp_col = (int*)malloc(sizeof(int)*q*2);
    for(i = 0;i < q;i++)
    {
        char c;
        while(fscanf(fp1,"%c",&c))
        {
          if(c == 'U')
            break;
        }
        while(fscanf(fp1,"%c",&c))
        {
          if(c == 'C')
            break;
        }
        int col_num,val,p;
        char op_type;
        fscanf(fp1,"%d%d%d",&col_num,&val,&p);
        comp_col[2*i] = col_num;
        comp_col[2*i+1] = val;
        for(j = 0;j < p;j++)
        {
            int col_to_upd,upd_val;
            while(fscanf(fp1,"%c",&c))
            {
              if(c == 'C')
                break;
            }
            fscanf(fp1,"%d%d",&col_to_upd,&upd_val);
            while(fscanf(fp1,"%c",&op_type))
            {
                if((op_type == '+') || (op_type == '-'))
                  break;
            }
            if(op_type ==  '-')
              upd_val *= -1;
            upd_vals[20*i+j] = upd_val;
            col_vals[20*i+j] = col_to_upd;
        }
        for(j = p;j < 20;j++)
        {
            col_vals[20*i+j] = 0;//no update operations
        }
    }
    int *k1;
    int *k2;
    int* arr1;
    int* col_v;
    hipMalloc(&k1,sizeof(int)*q*20);
    hipMalloc(&k2,sizeof(int)*q*20);
    hipMalloc(&arr1,sizeof(int)*m*n);
    hipMalloc(&col_v,sizeof(int)*q*2);
    hipMemcpy(col_v,comp_col,sizeof(int)*q*2,hipMemcpyHostToDevice);
    hipMemcpy(arr1,arr,sizeof(int)*m*n,hipMemcpyHostToDevice);
    hipMemcpy(k1,upd_vals,sizeof(int)*q*20,hipMemcpyHostToDevice);
    hipMemcpy(k2,col_vals,sizeof(int)*q*20,hipMemcpyHostToDevice);
    int x = ceil((m*q*20.0)/1024);
    database_update<<<x,1024>>>(arr1,col_v,k2,k1,m,n,q);
    hipDeviceSynchronize();
    hipMemcpy(arr,arr1,sizeof(int)*m*n,hipMemcpyDeviceToHost);
    for(i = 0;i < m;i++)
    {
        for(j = 0;j < n;j++)
        {
            fprintf(fp2,"%d ",arr[n*i+j]);
        }
        fprintf(fp2,"\n");
    }
    fclose(fp1);
    fclose(fp2);
    return 0;
}
