#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>
__global__ void sumRandC(int *a,int* b,int m,int n,int p,int q,int k)
{
 int beg_id = k*(blockDim.x*blockIdx.x + threadIdx.x);
 int i = 0;
 while((i < k) && (beg_id < m*n))
 {
     i++;
    int r = beg_id / n;
    int c = beg_id % n;
    int x = a[beg_id];
    int *ptr1 = b + (q*(r+1)-1);
    int *ptr2 = b + (q*(p-1) + c);
    atomicAdd(ptr1,x);
    atomicAdd(ptr2,x);
    beg_id++;
 }
}

__global__ void findMin(int *a,int *b,int m,int n,int p,int q,int k)
{
 int beg_id = k*(blockDim.x*blockIdx.x + threadIdx.x);
 int i = 0;
 int *ptr = &b[p*q-1];
 while((i < k) && (beg_id < m*n))
 {
    int r = beg_id / n;
    int c = beg_id % n;
    if((r == m-1) || (c == n-1))
    {
        if(r == m-1)
          r++;
        else
          c++;
      int x = b[r*q + c];
      atomicMin(ptr,x);
    }
    i++;
    beg_id++;
 }
}

__global__ void updateMin(int *a,int *b,int m,int n,int p,int q,int k)
{
 int beg_id = k*(blockDim.x*blockIdx.x + threadIdx.x);
 int i = 0;
 int mini = b[p*q-1];
 while((i < k) && (beg_id < m*n))
 {
     i++;
    int r = beg_id / n;
    int c = beg_id % n;
    b[r*q + c] += mini;
    beg_id++;
 }
}
int main(void)
{
    int m,n,k;
    scanf("%d%d%d",&m,&n,&k);
    int* arr = new int[m*n];
    int* arr1 = new int[(m+1)*(n+1)];
    int i,j;
    for(i = 0;i < m;i++)
    {
        for(j = 0;j < n;j++)
        {
          scanf("%d",&arr[n*i+j]);
          arr1[(n+1)*i + j] = arr[n*i+j];
        }
    }
    int t = (n+1)*m;
    for(i = 0;i <= n;i++)
    {
        arr1[t + i] = 0;
    }
    for(i = 0;i <= m;i++)
    {
        arr1[(n+1)*i + n] = 0;
    }
    arr1[(m+1)*(n+1)-1] = INT_MAX;
    int* a;
    hipMalloc(&a,n*m*sizeof(int));
    hipMemcpy(a,arr,n*m*sizeof(int),hipMemcpyHostToDevice);
    int* b;
    hipMalloc(&b,(n+1)*(m+1)*sizeof(int));
    hipMemcpy(b,arr1,(n+1)*(m+1)*sizeof(int),hipMemcpyHostToDevice);
    int tn = (m*n)/k;
    int bn = 1;
    if(tn > 1024)
    {
        bn = ceil(((double)tn)/1024);
        tn = 1024;
    }
    sumRandC<<<bn,tn>>>(a,b,m,n,m+1,n+1,k);
    hipDeviceSynchronize();
    findMin<<<bn,tn>>>(a,b,m,n,m+1,n+1,k);
    hipDeviceSynchronize();
    updateMin<<<bn,tn>>>(a,b,m,n,m+1,n+1,k);
    hipDeviceSynchronize();
    hipMemcpy(arr1,b,(n+1)*(m+1)*sizeof(int),hipMemcpyDeviceToHost);
     for(i = 0;i <= m;i++)
     {
         for(j = 0;j <= n;j++)
         {
             printf("%d ",arr1[(n+1)*i+j]);
         }
        printf("\n");
     }
    return 0;
}